#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "memory_manager.cuh"

// Test kernels
__global__ void writeTest(int* data, int size, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = value;
    }
}

__global__ void verifyTest(int* data, int size, int value, int* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && data[idx] != value) {
        atomicAdd(result, 1);
    }
}

// Main test function
int main() {
    printf("Starting memory manager tests...\n");

    // Initialize memory manager with 1GB
    if (!initializeMemoryManager(1ULL * 1024 * 1024 * 1024)) {
        printf("Failed to initialize memory manager\n");
        return 1;
    }

    // Test basic allocation
    size_t alloc_size = 256 * 1024 * 1024; // 256MB
    void* ptr = allocateMemory(alloc_size);
    if (!ptr) {
        printf("Failed to allocate memory\n");
        return 1;
    }

    // Test kernel execution with allocated memory
    int num_elements = alloc_size / sizeof(int);
    int threads = 256;
    int blocks = (num_elements + threads - 1) / threads;
    int test_value = 42;

    // Write test
    writeTest<<<blocks, threads>>>((int*)ptr, num_elements, test_value);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Write kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipDeviceSynchronize();

    // Verify test
    int* d_result;
    hipMalloc(&d_result, sizeof(int));
    hipMemset(d_result, 0, sizeof(int));

    verifyTest<<<blocks, threads>>>((int*)ptr, num_elements, test_value, d_result);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Verify kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_result);
        return 1;
    }
    hipDeviceSynchronize();

    // Check result
    int h_result;
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);

    if (h_result != 0) {
        printf("Memory verification failed: %d errors found\n", h_result);
        return 1;
    }

    // Test memory info
    size_t available = getAvailableMemory();
    printf("Available memory: %zu bytes\n", available);

    // Test memory deallocation
    if (!freeMemory(ptr)) {
        printf("Failed to free memory\n");
        return 1;
    }

    // Test memory defragmentation
    if (!defragmentMemory()) {
        printf("Failed to defragment memory\n");
        return 1;
    }

    // Cleanup
    if (!shutdownMemoryManager()) {
        printf("Failed to shutdown memory manager\n");
        return 1;
    }

    printf("All tests passed successfully!\n");
    return 0;
}
